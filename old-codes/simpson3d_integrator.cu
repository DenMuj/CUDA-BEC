#include "hip/hip_runtime.h"
// simpson3d_integrator.cu - Modified version with device memory support
#include "simpson3d_integrator.hpp"
#include "simpson3d_kernel.cuh"
#include <hip/hip_runtime.h>
#include <iostream>
#include <algorithm>

// Implementation class (hidden from public interface)
class Simpson3DTiledIntegratorImpl {
public:
    double *d_f;           // Device memory for current tile
    double *d_partial_sum; // Device memory for accumulating results
    double *h_tile_sum_pinned; // Pinned host memory for async transfers
    hipStream_t stream;   // CUDA stream for async operations
    long tile_size_z;      // Number of z-slices per tile
    long max_tile_points;  // Maximum points in a tile
    long cached_Nx;        // Cached grid dimensions
    long cached_Ny;
    
    Simpson3DTiledIntegratorImpl(long Nx, long Ny, long tile_z) 
        : tile_size_z(tile_z), cached_Nx(Nx), cached_Ny(Ny) {
        
        // Allocate memory for one tile
        max_tile_points = Nx * Ny * tile_size_z;
        hipMalloc(&d_f, max_tile_points * sizeof(double));
        hipMalloc(&d_partial_sum, sizeof(double));
        
        // Allocate pinned host memory for async transfers
        hipHostAlloc(&h_tile_sum_pinned, sizeof(double), hipHostMallocDefault);
        
        // Create CUDA stream for async operations
        hipStreamCreate(&stream);
        
        // Check for allocation errors
        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            std::cerr << "CUDA allocation error: " << hipGetErrorString(err) << std::endl;
            throw std::runtime_error("Failed to allocate GPU memory");
        }
    }
    
    ~Simpson3DTiledIntegratorImpl() {
        hipFree(d_f);
        hipFree(d_partial_sum);
        hipHostFree(h_tile_sum_pinned);
        hipStreamDestroy(stream);
    }
    
    // Original function - copies from host memory
    double integrate(double hx, double hy, double hz, double *h_f, 
                    long Nx, long Ny, long Nz) {
        double total_sum = 0.0;
        
        // Process the volume in tiles along the Z direction
        for (long z_start = 0; z_start < Nz; z_start += tile_size_z) {
            // Calculate the actual size of this tile (last tile might be smaller)
            long current_tile_z = std::min(tile_size_z, Nz - z_start);
            long tile_points = Nx * Ny * current_tile_z;
            
            // Copy this tile's data from host to device
            hipMemcpy(d_f, h_f + z_start * Nx * Ny, 
                      tile_points * sizeof(double), hipMemcpyHostToDevice);
            
            // Reset the partial sum for this tile
            hipMemset(d_partial_sum, 0, sizeof(double));
            
            // Launch kernel for this tile
            launchSimpson3DKernel(d_f, d_partial_sum, Nx, Ny, Nz, 
                                 tile_size_z, z_start, current_tile_z);
            
            // Check for kernel launch errors
            hipError_t err = hipGetLastError();
            if (err != hipSuccess) {
                std::cerr << "Kernel launch error: " << hipGetErrorString(err) << std::endl;
                return 0.0;
            }
            
            // Check for kernel execution errors
            err = hipGetLastError();
            if (err != hipSuccess) {
                std::cerr << "Kernel execution error: " << hipGetErrorString(err) << std::endl;
                return 0.0;
            }
            
            // Use async transfer with pinned memory for better performance
            hipMemcpyAsync(h_tile_sum_pinned, d_partial_sum, sizeof(double), 
                           hipMemcpyDeviceToHost, stream);
            
            // Wait for the async transfer to complete
            hipStreamSynchronize(stream);
            
            // Accumulate the result
            total_sum += *h_tile_sum_pinned;
        }
        
        // Apply Simpson's rule scaling factor
        return total_sum * hx * hy * hz / 27.0;
    }
    
    // New function - works with device memory directly
    double integrateDevice(double hx, double hy, double hz, double *d_f_full, 
                          long Nx, long Ny, long Nz) {
        double total_sum = 0.0;
        
        // Process the volume in tiles along the Z direction
        for (long z_start = 0; z_start < Nz; z_start += tile_size_z) {
            // Calculate the actual size of this tile (last tile might be smaller)
            long current_tile_z = std::min(tile_size_z, Nz - z_start);
            long tile_points = Nx * Ny * current_tile_z;
            
            // Copy this tile's data from device to device (GPU to GPU copy)
            hipMemcpy(d_f, d_f_full + z_start * Nx * Ny, 
                      tile_points * sizeof(double), hipMemcpyDeviceToDevice);
            
            // Reset the partial sum for this tile
            hipMemset(d_partial_sum, 0, sizeof(double));
            
            // Launch kernel for this tile
            launchSimpson3DKernel(d_f, d_partial_sum, Nx, Ny, Nz, 
                                 tile_size_z, z_start, current_tile_z);
            
            // Check for kernel launch errors
            hipError_t err = hipGetLastError();
            if (err != hipSuccess) {
                std::cerr << "Kernel launch error: " << hipGetErrorString(err) << std::endl;
                return 0.0;
            }
            
            // Check for kernel execution errors
            err = hipGetLastError();
            if (err != hipSuccess) {
                std::cerr << "Kernel execution error: " << hipGetErrorString(err) << std::endl;
                return 0.0;
            }
            
            // Use async transfer with pinned memory for better performance
            hipMemcpyAsync(h_tile_sum_pinned, d_partial_sum, sizeof(double), 
                           hipMemcpyDeviceToHost, stream);
            
            // Wait for the async transfer to complete
            hipStreamSynchronize(stream);
            
            // Accumulate the result
            total_sum += *h_tile_sum_pinned;
        }
        
        // Apply Simpson's rule scaling factor
        return total_sum * hx * hy * hz / 27.0;
    }
    
    void setTileSize(long new_tile_size) {
        if (new_tile_size <= 0) {
            throw std::invalid_argument("Tile size must be positive");
        }
        
        tile_size_z = new_tile_size;
        // Reallocate if new size is larger
        long new_max_points = cached_Nx * cached_Ny * tile_size_z;
        if (new_max_points > max_tile_points) {
            hipFree(d_f);
            max_tile_points = new_max_points;
            hipMalloc(&d_f, max_tile_points * sizeof(double));
            
            hipError_t err = hipGetLastError();
            if (err != hipSuccess) {
                throw std::runtime_error("Failed to reallocate GPU memory");
            }
        }
    }
};

// Public interface implementation
Simpson3DTiledIntegrator::Simpson3DTiledIntegrator(long Nx, long Ny, long tile_z) {
    pImpl = new Simpson3DTiledIntegratorImpl(Nx, Ny, tile_z);
}

Simpson3DTiledIntegrator::~Simpson3DTiledIntegrator() {
    delete pImpl;
}

// Host memory version
double Simpson3DTiledIntegrator::integrate(double hx, double hy, double hz, 
                                          double* h_f, long Nx, long Ny, long Nz) {
    return pImpl->integrate(hx, hy, hz, h_f, Nx, Ny, Nz);
}

// Device memory version
double Simpson3DTiledIntegrator::integrateDevice(double hx, double hy, double hz,
                                                double* d_f, long Nx, long Ny, long Nz) {
    return pImpl->integrateDevice(hx, hy, hz, d_f, Nx, Ny, Nz);
}

void Simpson3DTiledIntegrator::setTileSize(long new_tile_size) {
    pImpl->setTileSize(new_tile_size);
}

size_t Simpson3DTiledIntegrator::getMemoryUsage(long Nx, long Ny) const {
    return (Nx * Ny * pImpl->tile_size_z + 1) * sizeof(double);
}

long Simpson3DTiledIntegrator::getTileSize() const {
    return pImpl->tile_size_z;
}